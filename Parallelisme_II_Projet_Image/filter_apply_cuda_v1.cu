#include "hip/hip_runtime.h"
extern "C" {
#include "filter_lib.h"
#include "ppm_lib.h"
}

#define BLOCK_DIM 32

__global__ void kernelFilterConvolution(const PPMImage *inputImage, const PPMFilter *filter, PPMPixel *outputImageData)
{
	const int imageWidth = inputImage->w;
	const int imageHeight = inputImage->h;
	const int filterWidth = filter->w;
	const int filterHeight = filter->h;

	int pixelCoordX = blockIdx.x * blockDim.x + threadIdx.x;
	int pixelCoordY = blockIdx.y * blockDim.y + threadIdx.y;

	if((pixelCoordX >= imageWidth) || (pixelCoordY >= imageHeight))
		return;

	int halfW = (filterWidth - 1) >> 1;
	int halfH = (filterHeight - 1) >> 1;
	int k = imageWidth - 1;
	int l = imageHeight - 1;

	int sumR = 0;
	int sumG = 0;
	int sumB = 0;

	int divisionFactor = 0;

	for (int i = -halfW; i <= halfW; i++)
	{
		for (int j = -halfH; j <= halfH; j++)
		{
			int x = min(max(pixelCoordX + i, 0), k);
			int y = min(max(pixelCoordY + j, 0), l);
			int pixelIndex = y * imageWidth + x;
			PPMPixel currentPixel = inputImage->data[pixelIndex];

			int weightIndex = (halfW + i) * filterWidth + (halfH + j);
			int currentWeight = filter->data[weightIndex];

			divisionFactor += currentWeight;
			sumR += (currentPixel.r) * currentWeight;
			sumG += (currentPixel.g) * currentWeight;
			sumB += (currentPixel.b) * currentWeight;
		}
	}	

	PPMPixel newPixel;
	if (divisionFactor == 0) divisionFactor = 1;
	newPixel.r = (Uint8)((float)sumR / (float)divisionFactor);
	newPixel.g = (Uint8)((float)sumG / (float)divisionFactor);
	newPixel.b = (Uint8)((float)sumB / (float)divisionFactor);

	outputImageData[pixelCoordY * imageWidth + pixelCoordX] = newPixel;
}

extern "C"
void applyFilter(PPMImage *image, PPMFilter *filter)
{
	dim3 gridDim;
	gridDim.x = (int)((float)image->w / (float)BLOCK_DIM) + 1;
	gridDim.y = (int)((float)image->h / (float)BLOCK_DIM) + 1;
	gridDim.z = 1;

	dim3 blockDim;
	blockDim.x = BLOCK_DIM;
	blockDim.y = BLOCK_DIM;
	blockDim.z = 1;

	PPMImage *deviceInputImage;
	PPMPixel *deviceInputImageData;
	PPMFilter *deviceFilter;
	int* deviceFilterData;
	PPMPixel *deviceOutputImageData;

	int* originalFilterDataPtr = filter->data;
	PPMPixel* originalImageDataPtr = image->data;

	int imageDataSize = image->w*image->h*sizeof(PPMPixel);
	int filterDataSize = filter->w*filter->h*sizeof(int);
	
	hipMalloc(&deviceInputImageData, imageDataSize);
	hipMemcpy(deviceInputImageData, image->data, imageDataSize, hipMemcpyHostToDevice);
	hipMalloc(&deviceInputImage, sizeof(PPMImage));
	image->data = deviceInputImageData;
	hipMemcpy(deviceInputImage, image, sizeof(PPMImage), hipMemcpyHostToDevice);
	
	hipMalloc(&deviceFilterData, filterDataSize);
	hipMemcpy(deviceFilterData, filter->data, filterDataSize, hipMemcpyHostToDevice);
	hipMalloc(&deviceFilter, sizeof(PPMFilter));
	filter->data = deviceFilterData;
	hipMemcpy(deviceFilter, filter, sizeof(PPMFilter), hipMemcpyHostToDevice);
	filter->data = originalFilterDataPtr;

	hipMalloc(&deviceOutputImageData, imageDataSize);

	kernelFilterConvolution <<<gridDim, blockDim >>> (deviceInputImage, deviceFilter, deviceOutputImageData);
	hipDeviceSynchronize();

	image->data = originalImageDataPtr;
	hipMemcpy(image->data, deviceOutputImageData, imageDataSize, hipMemcpyDeviceToHost);

	hipFree(deviceInputImageData);
	hipFree(deviceInputImage);
	hipFree(deviceFilterData);
	hipFree(deviceFilter);
	hipFree(deviceOutputImageData);
}
