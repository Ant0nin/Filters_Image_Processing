#include "hip/hip_runtime.h"
extern "C" {
#include "filter_lib.h"
#include "ppm_lib.h"
}

#define BLOCK_WIDTH 32

__global__ void kernelFilterConvolution(const PPMImage *inputImage, const PPMFilter *filter, PPMPixel *outputImageData)
{
	const int imageWidth = inputImage->w;
	const int imageHeight = inputImage->h;
	const int filterWidth = filter->w;
	const int filterHeight = filter->h;

	int pixelCoordX = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelCoordY = blockIdx.x * blockDim.x + threadIdx.x;

	if((pixelCoordX >= imageWidth) || (pixelCoordY >= imageHeight))
		return;

	int halfW = (filterWidth - 1) >> 1;
	int halfH = (filterHeight - 1) >> 1;
	int k = imageWidth - 1;
	int l = imageHeight - 1;

	int sumR = 0;
	int sumG = 0;
	int sumB = 0;

	int divisionFactor = 0;

	for (int i = -halfW; i <= halfW; i++)
	{
		for (int j = -halfH; j <= halfH; j++)
		{
			int x = min(max(pixelCoordX + i, 0), k);
			int y = min(max(pixelCoordY + j, 0), l);
			int pixelIndex = y * imageWidth + x;
			PPMPixel currentPixel = inputImage->data[pixelIndex];

			int weightIndex = (halfW + i) * filterWidth + (halfH + j);
			int currentWeight = filter->data[weightIndex];

			divisionFactor += currentWeight;
			sumR += (currentPixel.r) * currentWeight;
			sumG += (currentPixel.g) * currentWeight;
			sumB += (currentPixel.b) * currentWeight;
		}
	}	

	PPMPixel newPixel;
	newPixel.r = (Uint8)(sumR / divisionFactor);
	newPixel.g = (Uint8)(sumG / divisionFactor);
	newPixel.b = (Uint8)(sumB / divisionFactor);

	outputImageData[pixelCoordY * imageWidth + pixelCoordX] = newPixel;
}

extern "C"
void applyFilter(PPMImage *image, PPMFilter *filter)
{
	dim3 gridDim;
	gridDim.x = (int)((float)image->w / (float)BLOCK_WIDTH); //+1;
	gridDim.y = (int)((float)image->h / (float)BLOCK_WIDTH); //+1;
	gridDim.z = 1;

	dim3 blockDim;
	blockDim.x = BLOCK_WIDTH;
	blockDim.y = BLOCK_WIDTH;
	blockDim.z = 1;

	PPMImage *deviceInputImage;
	PPMPixel *deviceInputImageData;
	PPMFilter *deviceFilter;
	int* deviceFilterData;
	PPMPixel *deviceOutputImageData;

	int* originalFilterDataPtr = filter->data;
	PPMPixel* originalImageDataPtr = image->data;

	int imageDataSize = image->w*image->h*sizeof(PPMPixel);
	int filterDataSize = filter->w*filter->h*sizeof(int);
	
	hipError_t check;

	check = hipMalloc(&deviceInputImageData, imageDataSize);
	check = hipMemcpy(deviceInputImageData, image->data, imageDataSize, hipMemcpyHostToDevice);
	check = hipMalloc(&deviceInputImage, sizeof(PPMImage));
	image->data = deviceInputImageData;
	check = hipMemcpy(deviceInputImage, image, sizeof(PPMImage), hipMemcpyHostToDevice);
	
	check = hipMalloc(&deviceFilterData, filterDataSize);
	check = hipMemcpy(deviceFilterData, filter->data, filterDataSize, hipMemcpyHostToDevice);
	check = hipMalloc(&deviceFilter, sizeof(PPMFilter));
	filter->data = deviceFilterData;
	check = hipMemcpy(deviceFilter, filter, sizeof(PPMFilter), hipMemcpyHostToDevice);
	filter->data = originalFilterDataPtr;

	check = hipMalloc(&deviceOutputImageData, imageDataSize);

	kernelFilterConvolution <<<gridDim, blockDim >>> (deviceInputImage, deviceFilter, deviceOutputImageData);
	hipDeviceSynchronize();

	image->data = originalImageDataPtr;
	check = hipMemcpy(image->data, deviceOutputImageData, imageDataSize, hipMemcpyDeviceToHost);

	check = hipFree(deviceInputImageData);
	check = hipFree(deviceInputImage);
	check = hipFree(deviceFilterData);
	check = hipFree(deviceFilter);
	check = hipFree(deviceOutputImageData);
}
